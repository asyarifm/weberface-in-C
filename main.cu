#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: Mar 17, 2016
 *      Author: mimos
 */

#include <stdio.h>
#include <opencv/cxcore.h>
#include <opencv/cv.h>
#include <opencv/highgui.h>
#include <opencv/cvaux.h>
#include <math.h>

CvMat* weberface(CvMat* image);
CvMat* loadGaborFilter(char path[100],int GaborH, int GaborW);
CvMat* minmaxNormalization(CvMat* image, int min, int max);
CvMat* rankNormalization(CvMat* image);
void quicksort(int* x, int* ind, int first,int last);

int main()
{
	int i, j;
	for (i = 1; i <= 38; i++)
	{
		for (j = 1; j <= 60; j++)
		{
			char f1[100];
			sprintf(f1, "CroppedYaleB/yale (%d)/%d (%d).png", i, i, j); //image path
			printf("load %s \n", f1);
			CvMat* img = cvLoadImageM(f1, CV_LOAD_IMAGE_GRAYSCALE); //load image
			printf("finish load %s \n", f1);

			//resize image
			CvMat* imgresize = cvCreateMat(img->rows/2, img->cols/2, img->type);
			cvResize(img, imgresize);
			printf("resized image \n");

			imgresize = minmaxNormalization(imgresize, 0, 255); //minmaxnormalization

			CvMat* result = weberface(imgresize);  //weberface implementation

			result = rankNormalization(result); //minmaxnormalization


			char f2[100];
			sprintf(f2, "CroppedYaleB/yale (%d)/after weberface/%d (%d).png", i, i, j);
			printf("Save %s \n", f2);
			cvSaveImage(f2, result);
			printf("finish save %s \n", f1);

			cvReleaseMat(&result);
			cvReleaseMat(&img);
			cvReleaseMat(&imgresize);
		}
	}

	return 0;
}

/* Function: minmaxNormalization
 *
 *
 *
 */
CvMat* minmaxNormalization(CvMat* image, int min, int max)
{
	int i, j;
	CvMat* result = cvCreateMat(image->rows, image->cols, CV_32FC1);
	cvNormalize(image, result, min, max, CV_MINMAX);

	for (i = 0; i < result->rows; i++)
	{
		for (j = 0; j < result->cols; j++)
		{
			CvScalar s;
			s = cvGet2D(result, i, j);
			s.val[0] = ceil(s.val[0]);
			cvSet2D(result, i, j, s);
		}
	}

	return result;
}

/* Function: weberface
 *
 *
 */
CvMat* weberface(CvMat* image)
{
	int nn = 9, alfa = 2, sigma = 1;

	int in_one_dim = (sqrt(nn) - 1) / 2;

	int i = 0, j = 0, k = 0, l = 0;
	double sum = 0, argument = 0;

	image = minmaxNormalization(image, 0, 255);

	CvMat* padBlock = cvCreateMat(image->rows+(2*in_one_dim), image->cols+(2*in_one_dim), CV_32FC1);
	CvMat* result = cvCreateMat(image->rows, image->cols, CV_32FC1);
	CvMat* filter = loadGaborFilter("filter.txt", (2*ceil(3*sigma))+1, (2*ceil(3*sigma))+1);  		//loadfilter for gaussian
	cvFilter2D(image, image, filter, cvPoint(-1,-1));										  		//Gaussian FIltering

	//create replicate padding to the block
	for(i = 0; i < padBlock->rows; i++)
	{
		for(j = 0; j < padBlock->cols; j++)
		{
			CvScalar s;
			if (i == 0)
			{
				if (j == 0)
				{
					s = cvGet2D(image, i, j);
				}
				else if (j == padBlock->cols - 1)
				{
					s = cvGet2D(image, i, j - 2);
				}
				else
				{
					s = cvGet2D(image, i, j - 1);
				}
			}
			else if (i == padBlock->rows - 1)
			{
				if (j == 0)
				{
					s = cvGet2D(image, i - 2, j);
				}
				else if (j == padBlock->cols - 1)
				{
					s = cvGet2D(image, i - 2, j - 2);
				}
				else
				{
					s = cvGet2D(image, i - 2, j - 1);
				}
			}
			else if (j == 0)
			{
				s = cvGet2D(image, i - 1, j);
			}
			else if (j == padBlock->cols - 1)
			{
				s = cvGet2D(image, i - 1, j - 2);
			}
			else
			{
				s = cvGet2D(image, i - 1, j - 1);
			}
			cvSet2D(padBlock, i, j, s);
		}
	}

	for(i = 1; i < padBlock->rows - 1; i++)
	{
		for (j = 1; j < padBlock->cols - 1; j++)
		{
			sum = 0;
			CvScalar s;
			s = cvGet2D(padBlock, i, j);

			for(k = i - 1; k <= i + 1; k++)
			{
				for(l = j - 1; l <= j + 1; l++)
				{
					CvScalar t;
					t = cvGet2D(padBlock, k, l);
					argument = (s.val[0] - t.val[0])/(s.val[0]+0.01);
					sum = sum + argument;
				}
			}

			CvScalar v;
			v.val[0] = atan(alfa * sum);
			cvSet2D(result, i - 1, j - 1, v);
		}
	}

	result = minmaxNormalization(result, 0, 255);

	//clear memory
	cvReleaseMat(&padBlock);

	return result;
}

/* Function: rankNormalization
 *
 *
 *
 */
CvMat* rankNormalization(CvMat* image)
{
	int i, j, count = 0;
	int N = image->cols * image->rows;
	int singleColumn[N];
	int index[N];
	CvMat* result = cvCreateMat(image->rows, image->cols, CV_32FC1);

	result = minmaxNormalization(image, 0, 255);

	for(i = 0; i < result->cols; i++)
	{
		for(j = 0; j < result->rows; j++)
		{
			CvScalar s;
			s = cvGet2D(result, j, i);

			singleColumn[count] = (int) s.val[0];
			index[count] = count;
			count = count + 1;
		}
	}

	quicksort(singleColumn, index, 0, N);

	for(i = 0; i < N; i++)
	{
		singleColumn[index[i]] = i + 1;
	}

	count = 0;
	for(i = 0; i < result->cols; i++)
	{
		for(j = 0; j < result->rows; j++)
		{
			CvScalar s;
			s.val[0] = singleColumn[count];
			cvSet2D(result, j, i, s);
			count = count + 1;
		}
	}

	result = minmaxNormalization(result, 0, 255);
	return result;
}

CvMat* loadGaborFilter(char path[100],int GaborH, int GaborW)
{
	CvMat* img = cvCreateMat(GaborH, GaborW, CV_32FC1);

	FILE *fileReal = fopen(path, "r");

	if (fileReal == NULL)
	{
		printf("\'%s\' is not found!\n", path);
		exit(1);
	}

	double var[GaborH][GaborW];
	int i = 0, j = 0;

	while(!feof(fileReal))
	{
		for(i = 0; i < GaborH; i++)
		{
			for(j = 0; j < GaborW; j++)
			{
				fscanf(fileReal, "%le", &var[i][j]);
			}
		}
	}

	fclose(fileReal);

	//convert 2D Array -> CvMat
	for(i = 0; i < GaborH; i++)
	{
		for(j = 0; j < GaborW; j++)
		{
			CvScalar s;
			s.val[0]=var[i][j];
			cvSet2D(img, i, j, s);
		}
	}

	return img;
}

void quicksort(int* x, int* ind, int first,int last)
{
    int pivot, i, j, itemp, temp;

     if(first < last)
     {
         pivot = first;
         i = first;
         j = last;

         while(i < j)
         {
             while(x[i] <= x[pivot] && i < last)
                 i++;
             while(x[j] > x[pivot])
                 j--;
             if(i < j)
             {
            	 temp = x[i];
            	 x[i] = x[j];
            	 x[j] = temp;

            	 itemp = ind[i];
            	 ind[i] = ind[j];
            	 ind[j] = itemp;
             }
         }

         temp = x[pivot];
         x[pivot] = x[j];
         x[j] = temp;

         itemp = ind[pivot];
         ind[pivot] = ind[j];
         ind[j] = itemp;

         quicksort(x, ind, first, j-1);
         quicksort(x, ind, j+1, last);
    }
}
